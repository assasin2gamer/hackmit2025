#include "hip/hip_runtime.h"
#include "sponge.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <fstream>
#include <iostream>
#include <vector>
#include <random>
#include <cmath>


#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    }

__global__ void insert_new_returns(float* buffer, const float* new_row,
                                   int N, int window, int t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int row = t % window;
        buffer[row * N + idx] = new_row[idx];
    }
}

__global__ void update_sums(const float* new_row, const float* old_row,
                            float* sum, float* sum2, float* sum_xy,
                            int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float xn = new_row[i];
        float xo = old_row[i];

                sum[i]  += xn - xo;
        sum2[i] += xn * xn - xo * xo;

        for (int j = 0; j < N; j++) {
            float yn = new_row[j];
            float yo = old_row[j];
            sum_xy[i * N + j] += xn * yn - xo * yo;
        }
    }
}

__global__ void corr_from_sums(const float* sum, const float* sum2,
                               const float* sum_xy, float* corr,
                               int N, int window) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < N && i <= j) {
        float mean_x = sum[i] / window;
        float mean_y = sum[j] / window;

        float cov   = (sum_xy[i * N + j] / window) - mean_x * mean_y;
        float var_x = (sum2[i] / window) - mean_x * mean_x;
        float var_y = (sum2[j] / window) - mean_y * mean_y;

        float denom = sqrtf(var_x * var_y + 1e-8f);
        float val   = cov / denom;

        corr[i * N + j] = corr[j * N + i] = val;
    }
}

__global__ void split_pos_neg(const float* corr, float* A_pos, float* A_neg,
                              int N, float eps) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < N) {
        float val = corr[i * N + j];
        if (val >= 0) {
            A_pos[i * N + j] = val;
            A_neg[i * N + j] = 0.0f;
        } else {
            A_pos[i * N + j] = 0.0f;
            A_neg[i * N + j] = -val;
        }
        if (i == j) {
            A_pos[i * N + j] += eps;
            A_neg[i * N + j] += eps;
        }
    }
}

__global__ void row_sums(const float* A, float* D, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float sum = 0.0f;
        for (int j = 0; j < N; j++) sum += A[i * N + j];
        D[i] = sum;
    }
}

__global__ void normalize_laplacian(const float* A, const float* D, float* L, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < N) {
        float dij = (i == j ? D[i] : 0.0f) - A[i * N + j];
        float norm_i = 1.0f / sqrtf(D[i] + 1e-8f);
        float norm_j = 1.0f / sqrtf(D[j] + 1e-8f);
        L[i * N + j] = norm_i * dij * norm_j;
    }
}

void power_method(float* d_mat, float* d_vecs, int N, int k, int iters) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    thrust::device_vector<float> d_tmp(N);
    std::mt19937 gen(42);
    std::uniform_real_distribution<float> dist(-1, 1);

    for (int v = 0; v < k; v++) {
        thrust::host_vector<float> h_x(N);
        for (int i = 0; i < N; i++) h_x[i] = dist(gen);
        thrust::device_vector<float> d_x = h_x;

        for (int it = 0; it < iters; it++) {
            float alpha = 1.0f, beta = 0.0f;
            hipblasSgemv(handle, HIPBLAS_OP_N, N, N,
                        &alpha, d_mat, N,
                        thrust::raw_pointer_cast(d_x.data()), 1,
                        &beta, thrust::raw_pointer_cast(d_tmp.data()), 1);

            float norm;
            hipblasSnrm2(handle, N, thrust::raw_pointer_cast(d_tmp.data()), 1, &norm);
            float inv_norm = 1.0f / (norm + 1e-8f);
            hipblasSscal(handle, N, &inv_norm, thrust::raw_pointer_cast(d_tmp.data()), 1);

            d_x = d_tmp;
        }
        hipMemcpy(d_vecs + v * N, thrust::raw_pointer_cast(d_x.data()),
                   N * sizeof(float), hipMemcpyDeviceToDevice);
    }
    hipblasDestroy(handle);
}


__global__ void assign_clusters(const float* X, const float* centroids,
                                int* labels, int N, int k, int dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float best_dist = 1e30;
        int best_cluster = 0;
        for (int c = 0; c < k; c++) {
            float dist = 0.0f;
            for (int d = 0; d < dim; d++) {
                float diff = X[i * dim + d] - centroids[c * dim + d];
                dist += diff * diff;
            }
            if (dist < best_dist) {
                best_dist = dist;
                best_cluster = c;
            }
        }
        labels[i] = best_cluster;
    }
}

__global__ void update_centroids(const float* X, float* centroids,
                                 const int* labels, int N, int k, int dim) {
    int c = blockIdx.x;
    int d = threadIdx.x;
    if (c < k && d < dim) {
        float sum = 0.0f;
        int count = 0;
        for (int i = 0; i < N; i++) {
            if (labels[i] == c) {
                sum += X[i * dim + d];
                count++;
            }
        }
        if (count > 0) centroids[c * dim + d] = sum / count;
    }
}

void kmeans_gpu(float* d_X, int N, int dim, int k,
                int max_iter, std::vector<int>& h_labels) {
    thrust::device_vector<int> d_labels(N);
    thrust::device_vector<float> d_centroids(k * dim);

        thrust::host_vector<float> h_init(k * dim);
    std::mt19937 gen(42);
    std::uniform_real_distribution<float> dist(-1, 1);
    for (int i = 0; i < k * dim; i++) h_init[i] = dist(gen);
    d_centroids = h_init;

    for (int it = 0; it < max_iter; it++) {
        assign_clusters<<<(N+255)/256, 256>>>(d_X,
                    thrust::raw_pointer_cast(d_centroids.data()),
                    thrust::raw_pointer_cast(d_labels.data()),
                    N, k, dim);
        update_centroids<<<k, dim>>>(d_X,
                    thrust::raw_pointer_cast(d_centroids.data()),
                    thrust::raw_pointer_cast(d_labels.data()),
                    N, k, dim);
    }

    h_labels.resize(N);
    hipMemcpy(h_labels.data(), thrust::raw_pointer_cast(d_labels.data()),
               N * sizeof(int), hipMemcpyDeviceToHost);
}


void spongesym_live_gpu(int N, int window, int k, int ticks,
                        const std::string& out_file, bool test) {
    float eps = 1e-3f;

        float *d_buffer, *d_corr, *A_pos, *A_neg, *D_pos, *D_neg, *L_pos, *L_neg, *M, *d_embedding;
    float *d_row[2], *d_old_row;
    float *d_sum, *d_sum2, *d_sum_xy;

    CUDA_CHECK(hipMalloc(&d_buffer, window * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_corr,   N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&A_pos,    N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&A_neg,    N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&D_pos,    N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&D_neg,    N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&L_pos,    N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&L_neg,    N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&M,        N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_embedding, N * k * sizeof(float)));

    CUDA_CHECK(hipMalloc(&d_row[0], N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_row[1], N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_old_row, N * sizeof(float)));

    CUDA_CHECK(hipMalloc(&d_sum,    N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_sum2,   N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_sum_xy, N * N * sizeof(float)));

    CUDA_CHECK(hipMemset(d_sum,  0, N * sizeof(float)));
    CUDA_CHECK(hipMemset(d_sum2, 0, N * sizeof(float)));
    CUDA_CHECK(hipMemset(d_sum_xy, 0, N * N * sizeof(float)));

        float* h_row[2];
    CUDA_CHECK(hipHostAlloc(&h_row[0], N * sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&h_row[1], N * sizeof(float), hipHostMallocDefault));

    std::mt19937 gen(42);
    std::normal_distribution<float> dist(0, 0.01);

    dim3 threads2D(16, 16);
    dim3 blocks2D((N+15)/16, (N+15)/16);

        hipStream_t stream_copy, stream_compute;
    CUDA_CHECK(hipStreamCreate(&stream_copy));
    CUDA_CHECK(hipStreamCreate(&stream_compute));

    int buf_id = 0;

    for (int t = 0; t < ticks; t++) {
        buf_id = t % 2;

                for (int i = 0; i < N; i++) h_row[buf_id][i] = dist(gen);

                CUDA_CHECK(hipMemcpyAsync(d_row[buf_id], h_row[buf_id],
                                   N * sizeof(float),
                                   hipMemcpyHostToDevice,
                                   stream_copy));

                if (t > 0) {
            int prev = (t - 1) % 2;
            int row = (t - 1) % window;

                        CUDA_CHECK(hipMemcpyAsync(d_old_row, d_buffer + row * N,
                                       N * sizeof(float),
                                       hipMemcpyDeviceToDevice,
                                       stream_compute));

                        insert_new_returns<<<(N+255)/256, 256, 0, stream_compute>>>(
                d_buffer, d_row[prev], N, window, t-1);

                        update_sums<<<(N+255)/256, 256, 0, stream_compute>>>(
                d_row[prev], d_old_row, d_sum, d_sum2, d_sum_xy, N);

                        corr_from_sums<<<blocks2D, threads2D, 0, stream_compute>>>(
                d_sum, d_sum2, d_sum_xy, d_corr, N, window);

                        split_pos_neg<<<blocks2D, threads2D, 0, stream_compute>>>(
                d_corr, A_pos, A_neg, N, eps);

                        row_sums<<<(N+255)/256, 256, 0, stream_compute>>>(A_pos, D_pos, N);
            row_sums<<<(N+255)/256, 256, 0, stream_compute>>>(A_neg, D_neg, N);
            normalize_laplacian<<<blocks2D, threads2D, 0, stream_compute>>>(A_pos, D_pos, L_pos, N);
            normalize_laplacian<<<blocks2D, threads2D, 0, stream_compute>>>(A_neg, D_neg, L_neg, N);

                        CUDA_CHECK(hipMemcpyAsync(M, L_pos, N * N * sizeof(float),
                                       hipMemcpyDeviceToDevice,
                                       stream_compute));
            power_method(M, d_embedding, N, k, 20);

                        std::vector<int> labels;
            kmeans_gpu(d_embedding, N, k, k, 10, labels);

                        if (!test) {
                std::ofstream file(out_file, std::ios::app);
                std::vector<float> h_embedding(N * k);
                hipMemcpyAsync(h_embedding.data(), d_embedding,
                                N * k * sizeof(float),
                                hipMemcpyDeviceToHost,
                                stream_compute);
                hipStreamSynchronize(stream_compute);

                for (int i = 0; i < N; i++) {
                    file << labels[i];
                    for (int d = 0; d < k; d++)
                        file << "," << h_embedding[i * k + d];
                    file << "\n";
                }
            }
        }

                hipStreamSynchronize(stream_copy);
    }

        hipStreamSynchronize(stream_compute);

        hipFree(d_buffer); hipFree(d_corr);
    hipFree(A_pos); hipFree(A_neg);
    hipFree(D_pos); hipFree(D_neg);
    hipFree(L_pos); hipFree(L_neg);
    hipFree(M); hipFree(d_embedding);
    hipFree(d_row[0]); hipFree(d_row[1]);
    hipFree(d_old_row);
    hipFree(d_sum); hipFree(d_sum2); hipFree(d_sum_xy);
    hipHostFree(h_row[0]); hipHostFree(h_row[1]);
    hipStreamDestroy(stream_copy); hipStreamDestroy(stream_compute);
}
